#include "hip/hip_runtime.h"
#include "LeNet5_cuda.h"

    /* NOTE: unless you want to make a major change to this class structure, 
    *  you need to write your output to the device memory d_output 
    *  so that classify() can handle the rest.
    */

__global__
void normalize(const uint8_t* const image, double* input) {
  //Initialize variables
  float max_int = 255.0L;
  float mean = 0.5L;
  float var = 0.5L;

  //i = 0 to batch*input_channel*input_size*input_size
  //each block have DimBlock size, and there are DimGrid blocks
  //ref: Cuda Thread indexing cheetsheet
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int i = blockId *(blockDim.x * blockDim.y) 
          + (threadIdx.y * blockDim.x) + threadIdx.x;
  
  input[i] = image[i] / max_int;
  input[i] = (input[i] - mean) / var;
}

__global__
void conv(double* input, double* output, double* weight,
          double* bias, int B, int H, int W, int IC, int OC,
          int K) {
  //Initialize variable
  //oc = blockIdx.x, b = blockIdx.y
  //w = threadIdx.x, h = threadIdx.y
  
  int output_index = blockIdx.y * (gridDim.x * blockDim.y * blockDim.x)
                    + blockIdx.x * (blockDim.y * blockDim.x)
                    + (int)fmaf(threadIdx.y,blockDim.x,threadIdx.x);
  double tmp = bias[blockIdx.x];

  #pragma unroll
  for(int ic = 0; ic < IC; ic++) {
    int input_base = blockIdx.y * (IC * H * W) + ic * (H * W) 
                  + (int)fmaf(threadIdx.y,W, threadIdx.x);
    int kernel_base = blockIdx.x * (IC * K * K) + ic * (K * K);

    //let out exit from loop due to threading.
    #pragma unroll
    for (int kh = 0; kh < K; kh++)
      for (int kw = 0; kw < K; kw++) {
        tmp += input[(int)fmaf(kh,W,input_base) + kw] *
              weight[(int)fmaf(kh,K,kernel_base) + kw];
    }
  }
  output[output_index] = tmp;
}

__global__
void relu(double* feature_map, int size) {
  //relu
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int i = blockId * (blockDim.x * blockDim.y)
            + (threadIdx.y * blockDim.x) + threadIdx.x;
  feature_map[i] = feature_map[i]>0.0 ? feature_map[i]:0.0;
  //feature_map[i] = fmaxf(feature_map[i], 0.0);
}

//Max Pooling
__global__
void pool(double* input, double* output, int B, int C, int H, int W) {

  int scale = 2;
  int H_OUT = H / scale;
  int W_OUT = W / scale;
  
  int input_base = blockIdx.y * (C * H * W)
                  + blockIdx.x * (H * W)
                  + (threadIdx.y*2) * (W)
                  + (threadIdx.x*2);
  //Init values
  double max_val = 0.0;

  //Find Maximum
  for(int sh = 0; sh < scale; sh++)
    for(int sw = 0; sw < scale; sw++) {
      double val = input[input_base + sh*(W) + sw];
      if(val > max_val) {
        max_val = val;
      }
    }

    int output_index = blockIdx.y * (C * H_OUT * W_OUT)
                    + blockIdx.x * (H_OUT * W_OUT)
                    + (threadIdx.y) * (W_OUT)
                    + (threadIdx.x);

    output[output_index] = max_val;
}

__global__
void fc(double* input, double* output, double* weight, double* bias,
        int B, int IC, int OC) {
  
  //Fully Connected
  int idx = (int)fmaf(blockIdx.x,blockDim.x,threadIdx.x);
  double tmp = bias[threadIdx.x];

  // __shared__ double sw[IC][IC];
  // sw[threadIdx.y][threadIdx.x] = weight[threadIdx.x*IC];

  // for(int ic = 0; ic < IC; ic++){
  //   tmp += sw[threadIdx.y][ic] * input[(int)fmaf(blockIdx.x,IC,ic)];
  // }
  #pragma unroll
  for(int ic = 0; ic < IC; ic++){
    tmp += weight[(int)fmaf(threadIdx.x, IC, ic)] * input[(int)fmaf(blockIdx.x,IC,ic)];
  }
  output[idx] = tmp;
}

void LeNet5_cuda::predict(int batch) {

    // TODO: conv1 - relu - pool1
    // TODO: conv2 - relu - pool2
    // TODO: fc1 - relu - fc2 - relu - fc3

    //Initialize grid and block dimensions
    dim3 DimGrid(input_channel, batch);
    dim3 DimBlock(input_size, input_size);

    //ToTensor and Normalize
    normalize<<<DimGrid, DimBlock>>> (d_image, d_input);
    hipDeviceSynchronize();

    // Conv2d
    DimGrid.x = conv1_out_channel; DimGrid.y = batch;
    DimBlock.x = input_size-(conv1_kernel_size-1); 
    DimBlock.y = input_size-(conv1_kernel_size-1);
    conv<<<DimGrid , DimBlock>>>(d_input, d_C1_feature_map, d_conv1_weight, 
           d_conv1_bias, batch, input_size, input_size, conv1_in_channel,
           conv1_out_channel, conv1_kernel_size);
    hipDeviceSynchronize();

    DimGrid.y = batch; DimGrid.x = C1_channel;
    DimBlock.y = C1_size; DimBlock.x = C1_size;
    relu<<<DimGrid, DimBlock>>>(d_C1_feature_map, batch * C1_channel * C1_size * C1_size);
    hipDeviceSynchronize();

    // MaxPool2d
    DimGrid.y = batch; DimGrid.x = C1_channel;
    DimBlock.y = C1_size/2; DimBlock.x = C1_size/2;
    pool<<<DimGrid, DimBlock>>>(d_C1_feature_map, d_S2_feature_map, batch, C1_channel, C1_size, C1_size);
    hipDeviceSynchronize();

    // Conv2d
    DimGrid.y = batch; DimGrid.x = conv2_out_channel;
    DimBlock.y = S2_size - (conv2_kernel_size - 1);
    DimBlock.x = S2_size - (conv2_kernel_size - 1);
    conv<<<DimGrid, DimBlock>>>(d_S2_feature_map, d_C3_feature_map, d_conv2_weight, d_conv2_bias, batch,
          S2_size, S2_size, conv2_in_channel, conv2_out_channel, conv2_kernel_size);
    hipDeviceSynchronize();

    DimGrid.y = batch; DimGrid.x = C3_channel;
    DimBlock.y = C3_size; DimBlock.x = C3_size;
    relu<<<DimGrid, DimBlock>>>(d_C3_feature_map, batch * C3_channel * C3_size * C3_size);
    hipDeviceSynchronize();

    // MaxPool2d
    DimGrid.y = batch; DimGrid.x = C3_channel;
    DimBlock.y = C3_size/2; DimBlock.x = C3_size/2;
    pool<<<DimGrid, DimBlock>>>(d_C3_feature_map, d_S4_feature_map, batch, 
          C3_channel, C3_size, C3_size);
    hipDeviceSynchronize();

    // Linear
    //DimGrid.y = 1; DimGrid.x = batch;
    //DimBlock.y = 1; DimBlock.x = fc1_out_channel;
    fc<<<batch, fc1_out_channel>>>(d_S4_feature_map, d_C5_layer, d_fc1_weight, d_fc1_bias,
                              batch, fc1_in_channel, fc1_out_channel);
    hipDeviceSynchronize();

    DimGrid.y = 1; DimGrid.x = batch;
    DimBlock.y = 1; DimBlock.x = C5_size;
    relu<<<DimGrid, DimBlock>>>(d_C5_layer, batch*C5_size);
    hipDeviceSynchronize();

    // Linear
    //DimGrid.y = batch; DimGrid.x = 1;
    //DimBlock.y = fc2_out_channel; DimBlock.x = 1;
    fc<<<batch, fc2_out_channel>>>(d_C5_layer, d_F6_layer, d_fc2_weight, d_fc2_bias,
                              batch, fc2_in_channel, fc2_out_channel);
    hipDeviceSynchronize();

    DimGrid.y = 1; DimGrid.x = batch;
    DimBlock.y = 1; DimBlock.x = F6_size;
    relu<<<DimGrid, DimBlock>>>(d_F6_layer, batch*F6_size);
    hipDeviceSynchronize();

    // Linear
    //DimGrid.y = batch; DimGrid.x = 1;
    //DimBlock.y = output_size; DimBlock.x = 1;
    fc<<<batch, output_size>>>(d_F6_layer, d_output, d_fc3_weight, d_fc3_bias,
                              batch, fc3_in_channel, fc3_out_channel);
    hipDeviceSynchronize();
}

void LeNet5_cuda::prepare_device_memory(uint8_t* image) {
  // Alloc Model Parameters
  hipMalloc((void**)&d_conv1_weight,
             sizeof(double) * conv1_in_channel * conv1_out_channel *
                 conv1_kernel_size * conv1_kernel_size);
  hipMalloc((void**)&d_conv1_bias, sizeof(double) * conv1_out_channel);
  hipMalloc((void**)&d_conv2_weight,
             sizeof(double) * conv2_in_channel * conv2_out_channel *
                 conv2_kernel_size * conv2_kernel_size);
  hipMalloc((void**)&d_conv2_bias, sizeof(double) * conv2_out_channel);
  hipMalloc((void**)&d_fc1_weight,
             sizeof(double) * fc1_in_channel * fc1_out_channel);
  hipMalloc((void**)&d_fc1_bias, sizeof(double) * fc1_out_channel);
  hipMalloc((void**)&d_fc2_weight,
             sizeof(double) * fc2_in_channel * fc2_out_channel);
  hipMalloc((void**)&d_fc2_bias, sizeof(double) * fc2_out_channel);
  hipMalloc((void**)&d_fc3_weight,
             sizeof(double) * fc3_in_channel * fc3_out_channel);
  hipMalloc((void**)&d_fc3_bias, sizeof(double) * fc3_out_channel);

  // Alloc Activations
  hipMalloc((void**)&d_image,
             sizeof(uint8_t) * batch * input_size * input_size * input_channel);
  hipMalloc((void**)&d_input,
             sizeof(double) * batch * input_channel * input_size * input_size);
  hipMalloc((void**)&d_C1_feature_map,
             sizeof(double) * batch * C1_channel * C1_size * C1_size);
  hipMalloc((void**)&d_S2_feature_map,
             sizeof(double) * batch * S2_channel * S2_size * S2_size);
  hipMalloc((void**)&d_C3_feature_map,
             sizeof(double) * batch * C3_channel * C3_size * C3_size);
  hipMalloc((void**)&d_S4_feature_map,
             sizeof(double) * batch * S4_channel * S4_size * S4_size);
  hipMalloc((void**)&d_C5_layer, sizeof(double) * batch * C5_size);
  hipMalloc((void**)&d_F6_layer, sizeof(double) * batch * F6_size);
  hipMalloc((void**)&d_output, sizeof(double) * batch * output_size);

  // Copy Parameters

  hipMemcpy(d_conv1_weight, conv1_weight,
             sizeof(double) * conv1_in_channel * conv1_out_channel *
                 conv1_kernel_size * conv1_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv1_bias, conv1_bias, sizeof(double) * conv1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_weight, conv2_weight,
             sizeof(double) * conv2_in_channel * conv2_out_channel *
                 conv2_kernel_size * conv2_kernel_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_conv2_bias, conv2_bias, sizeof(double) * conv2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc1_weight, fc1_weight,
             sizeof(double) * fc1_in_channel * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc1_bias, fc1_bias, sizeof(double) * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc2_weight, fc2_weight,
             sizeof(double) * fc2_in_channel * fc2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc2_bias, fc2_bias, sizeof(double) * fc2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc3_weight, fc3_weight,
             sizeof(double) * fc3_in_channel * fc3_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc3_bias, fc3_bias, sizeof(double) * fc3_out_channel,
             hipMemcpyHostToDevice);

  // copy input image
  size_t image_size = batch * input_size * input_size * input_channel;
  hipMemcpy(d_image, image, image_size * sizeof(uint8_t),
             hipMemcpyHostToDevice);

}

void LeNet5_cuda::classify(int* predict, int batch) {
  // read logits back to cpu
  hipMemcpy(output, d_output, sizeof(double) * output_size * batch,
             hipMemcpyDeviceToHost);
  // Softmax
  softmax(output, predict, batch, output_size);
}

LeNet5_cuda::~LeNet5_cuda() {
  hipFree(d_conv1_weight);   
  hipFree(d_conv2_weight);   
  hipFree(d_conv1_bias);     
  hipFree(d_conv2_bias);     
  hipFree(d_fc1_weight);     
  hipFree(d_fc2_weight);     
  hipFree(d_fc3_weight);     
  hipFree(d_fc1_bias);       
  hipFree(d_fc2_bias);       
  hipFree(d_fc3_bias);       

  hipFree(d_image);          
  hipFree(d_input);          
  hipFree(d_C1_feature_map); 
  hipFree(d_S2_feature_map); 
  hipFree(d_C3_feature_map); 
  hipFree(d_S4_feature_map); 
  hipFree(d_C5_layer);      
  hipFree(d_F6_layer);     
  hipFree(d_output);       
  hipFree(d_predict_cuda);   
}
